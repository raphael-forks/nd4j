
#include <hip/hip_runtime.h>
extern "C"
__global__ void mul_strided_double(int n, double *dx, double *dy,int incx,int incy) {
         int  dxIdx = blockDim.x * blockIdx.x + threadIdx.x;
         int dyIdx = blockDim.y * blockIdx.y + threadIdx.y;

        if(dxIdx < n && dxIdx % incx == 0 && dyIdx < n && dyIdx % incy == 0)
                 dy[dyIdx] *= dx[dxIdx];

 }

